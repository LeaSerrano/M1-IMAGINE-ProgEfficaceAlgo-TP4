
#include <hip/hip_runtime.h>
#include <iostream>


void vectorAddition(float* C, const float* A, const float* B, size_t n) {
        for (int i = 0; i < n; i++) {
            C[i] = A[i] + B[i];
        }
    } 

int main(int argc, char** argv){

    int n = 4;
    float *A, *B, *C;
    hipMallocManaged(&A, n * sizeof(float));
    hipMallocManaged(&B, n * sizeof(float));
    hipMallocManaged(&C, n * sizeof(float));

    A[0] = 1;
    A[1] = 2;
    A[2] = 3;
    A[3] = 4;

    B[0] = 4;
    B[1] = 3;
    B[2] = 2;
    B[3] = 1;

    vectorAddition(C, A, B, n);

    for (int i = 0; i < n; i++) {
        std::cout << C[i] << " " << std::endl;
    }

    hipFree(A);
    hipFree(B);
    hipFree(C);

}